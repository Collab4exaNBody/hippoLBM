#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid/domain_lbm.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <grid/lbm_parameters.hpp>
#include <hippoLBM/collision/bgk.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class CollisionBGK : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

      inline std::string documentation() const override final
      {
        return R"EOF( The `CollisionBGK` operator implements the Bhatnagar-Gross-Krook (BGK) collision model for the Lattice Boltzmann Method (LBM). This model assumes a single relaxation time approach  to approximate the collision process, driving the distribution functions toward equilibrium.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& traversals = *Traversals;
        auto& params = *Params;

        // define functor
        bgk<Q> func = {params.Fext};

        // get fields
        FieldView<3> pm1 = data.flux();
        int * const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        const double * const w = data.weights();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_data<Traversal::Real>();

        // run kernel
        parallel_for_id(ptr, size, func, parallel_execution_context(), pm1, pobst, pf, pm0, pex, pey, pez, w, params.tau);
      }
  };

  using CollisionBGK3D19Q = CollisionBGK<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(CollisionBGK)
  {
    OperatorNodeFactory::instance()->register_factory( "bgk", make_compatible_operator<CollisionBGK3D19Q>);
  }
}

