#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <grid/lbm_parameters.hpp>
#include <hippoLBM/collision/bgk.hpp>
#include <grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class CollisionBGK : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});
      ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);

      inline std::string documentation() const override final
      {
        return R"EOF( The `CollisionBGK` operator implements the Bhatnagar-Gross-Krook (BGK) collision model for the Lattice Boltzmann Method (LBM). This model assumes a single relaxation time approach  to approximate the collision process, driving the distribution functions toward equilibrium.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& traversals = *Traversals;
        auto& params = *Params;

        // get fields
        FieldView<3> pm1 = data.flux();
        int * const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        const double * const w = data.weights();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_levels();
        // define functor
        bgk<Q, Traversal::Real> func = {ptr, params.Fext, pm1, pobst, pf, pm0, pex, pey, pez, w, params.tau};
        // run kernel over the lbm grid
        parallel_for_simple(size, func, parallel_execution_context());
      }
  };

  using CollisionBGK3D19Q = CollisionBGK<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(CollisionBGK)
  {
    OperatorNodeFactory::instance()->register_factory( "bgk", make_variant_operator<CollisionBGK>);
  }
}

