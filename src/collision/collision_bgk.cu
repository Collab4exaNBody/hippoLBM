#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <grid_lbm/lbm_parameters.hpp>
#include <hipoLBM/collision/collision_bgk.hpp>

namespace hipoLBM
{
	using namespace onika;
	using namespace scg;
	using namespace onika::cuda;

	template<int Q>
		class CollisionBGQ : public OperatorNode
	{
		public:
			ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
			ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
			ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

			inline std::string documentation() const override final
			{
				return R"EOF( The `CollisionBGQ` operator implements the Bhatnagar-Gross-Krook (BGK) collision model for the Lattice Boltzmann Method (LBM). This model assumes a single relaxation time approach  to approximate the collision process, driving the distribution functions toward equilibrium.
        )EOF";
			}

			inline void execute () override final
			{
				auto& data = *GridDataQ;
				auto& traversals = *Traversals;
				auto& params = *Params;

				// define functor
				collision_bgk<Q> func = {params.Fext};

				// get fields
				math::Vec3d * const pm1 = data.flux();
				int * const pobst = data.obstacles();
				double * const pf = data.distributions();
				double * const pm0 = data.densities();
				const double * const w = data.weights();
				auto [pex, pey, pez] = data.exyz();

				// get traversal
				auto [ptr, size] = traversals.get_data<Traversal::Real>();

				// run kernel
				parallel_for_id(ptr, size, func, parallel_execution_context(), pm1, pobst, pf, pm0, pex, pey, pez, w, params.tau);
			}
	};

	using CollisionBGQ3D19Q = CollisionBGQ<19>;

	// === register factories ===  
	ONIKA_AUTORUN_INIT()
	{
		OperatorNodeFactory::instance()->register_factory( "collision_bgk", make_compatible_operator<CollisionBGQ3D19Q>);
	}
}

