#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <hippoLBM/grid/domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/grid/lbm_parameters.hpp>
#include <hippoLBM/collision/mrt.hpp>
#include <hippoLBM/grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class CollisionMRT : public OperatorNode
  {
    public:
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

      inline std::string documentation() const override final
      {
        return R"EOF( The `CollisionMRT` operator implements the MRT collision model for the Lattice Boltzmann Method (LBM). This model assumes a single relaxation time approach  to approximate the collision process, driving the distribution functions toward equilibrium.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *fields;
        auto& traversals = *Traversals;
        auto& params = *Params;

        // define functor
        mrt<Q> func = {params.Fext};

        // get fields
        int * const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        const double * const w = data.weights();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_data<Traversal::Real>();

        // run kernel
        parallel_for_id(ptr, size, func, parallel_execution_context(), pobst, pf, pm0, pex, pey, pez, w, params.tau);
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(CollisionMRT)
  {
    OperatorNodeFactory::instance()->register_factory( "mrt", make_variant_operator<CollisionMRT>);
  }
}

