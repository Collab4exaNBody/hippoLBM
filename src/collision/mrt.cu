#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <grid/lbm_parameters.hpp>
#include <hippoLBM/collision/mrt.hpp>
#include <grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class CollisionMRT : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

      inline std::string documentation() const override final
      {
        return R"EOF( The `CollisionMRT` operator implements the MRT collision model for the Lattice Boltzmann Method (LBM). This model assumes a single relaxation time approach  to approximate the collision process, driving the distribution functions toward equilibrium.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& traversals = *Traversals;
        auto& params = *Params;

        // define functor
        mrt<Q> func = {params.Fext};

        // get fields
        int * const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        const double * const w = data.weights();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_data<Traversal::Real>();

        // run kernel
        parallel_for_id(ptr, size, func, parallel_execution_context(), pobst, pf, pm0, pex, pey, pez, w, params.tau);
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(CollisionMRT)
  {
    OperatorNodeFactory::instance()->register_factory( "mrt", make_variant_operator<CollisionMRT>);
  }
}

