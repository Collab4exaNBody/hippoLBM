#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/parallel_for_core.cu>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/grid/lbm_parameters.hpp>
#include <hippoLBM/collision/mrt.hpp>
#include <hippoLBM/grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class CollisionMRT : public OperatorNode
  {
    public:
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

      inline std::string documentation() const override final
      {
        return R"EOF( The `CollisionMRT` operator implements the MRT collision model for the Lattice Boltzmann Method (LBM). This model assumes a single relaxation time approach  to approximate the collision process, driving the distribution functions toward equilibrium.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *fields;
        auto& traversals = *grid_region;
        auto& params = *Params;

        // define functor
        mrt<Q> func = {params.Fext};

        // get fields
        int * const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        const double * const w = data.weights();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_data<Traversal::Real>();

        // run kernel
        parallel_for_id(ptr, size, func, parallel_execution_context(), pobst, pf, pm0, pex, pey, pez, w, params.tau);
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(CollisionMRT)
  {
    OperatorNodeFactory::instance()->register_factory( "mrt", make_variant_operator<CollisionMRT>);
  }
}

