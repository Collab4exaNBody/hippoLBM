#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/compute/parallel_for_core.hpp>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/grid/lbm_parameters.hpp>
#include <hippoLBM/collision/macro_variables.hpp>

#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/update_ghost.hpp>
#include <hippoLBM/grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class MacroVariables : public OperatorNode
  {
    ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});
    public:

    inline std::string documentation() const override final
    {
      return R"EOF(  
      A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
      YAML example:
        - macro_variables
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *fields;
      auto& traversals = *grid_region;
      auto& params = *Params;

      // define functor
      macro_variables<Q> func = {params.Fext / 2};

      // get fields
      FieldView<3> pm1 = data.flux();
      int * const pobst = data.obstacles();
      FieldView<Q> pf = data.distributions();
      double * const pm0 = data.densities();
      auto [pex, pey, pez] = data.exyz();

      // get traversal
      auto [ptr, size] = traversals.get_data<Traversal::All>();

      // run kernel
      parallel_for_id(ptr, size, func, parallel_execution_context(), pm1, pobst, pf, pm0, pex, pey, pez);
    }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(macro_variables)
  {
    OperatorNodeFactory::instance()->register_factory( "macro_variables", make_variant_operator<MacroVariables>);
  }
}

