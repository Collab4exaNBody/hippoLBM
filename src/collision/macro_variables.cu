#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid/domain_lbm.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <grid/lbm_parameters.hpp>
#include <hippoLBM/collision/macro_variables.hpp>

#include <grid/domain_lbm.hpp>
#include <grid/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class MacroVariables : public OperatorNode
  {
      ADD_SLOT( lbm_fields<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

    public:

      inline std::string documentation() const override final
      {
        return R"EOF(  A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& traversals = *Traversals;
        auto& params = *Params;

        // define functor
        macro_variables<Q> func = {params.Fext / 2};

        // get fields
        FieldView<3> pm1 = data.flux();
        int * const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_data<Traversal::All>();

        // run kernel
        parallel_for_id(ptr, size, func, parallel_execution_context(), pm1, pobst, pf, pm0, pex, pey, pez);
      }
  };

  using MacroVariables3D19Q = MacroVariables<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(macro_variables)
  {
    OperatorNodeFactory::instance()->register_factory( "macro_variables", make_compatible_operator<MacroVariables3D19Q>);
  }
}

