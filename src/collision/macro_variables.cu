#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <grid_lbm/lbm_parameters.hpp>
#include <hippoLBM/collision/macro_variables.hpp>

#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class MacroVariables : public OperatorNode
  {
      ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});

    public:

      inline std::string documentation() const override final
      {
        return R"EOF(  A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& traversals = *Traversals;
        auto& params = *Params;

        // define functor
        macro_variables<Q> func = {params.Fext / 2};

        // get fields
        math::Vec3d * const pm1 = data.flux();
        int * const pobst = data.obstacles();
        WrapperF<Q> pf = data.distributions();
        double * const pm0 = data.densities();
        auto [pex, pey, pez] = data.exyz();

        // get traversal
        auto [ptr, size] = traversals.get_data<Traversal::All>();

        // run kernel
        parallel_for_id(ptr, size, func, parallel_execution_context(), pm1, pobst, pf, pm0, pex, pey, pez);
      }
  };

  using MacroVariables3D19Q = MacroVariables<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(macro_variables)
  {
    OperatorNodeFactory::instance()->register_factory( "macro_variables", make_compatible_operator<MacroVariables3D19Q>);
  }
}

