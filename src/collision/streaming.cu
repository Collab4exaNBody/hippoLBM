#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/parallel_for_box.hpp>
#include <grid_lbm/traversal_lbm.hpp>
#include <hipoLBM/collision/streaming.hpp>
#include <grid_lbm/update_ghost.hpp>

namespace hipoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;



  template<int Q>
    class StreamingLBM : public OperatorNode
  {
    public:
      ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);
      ADD_SLOT( bool, asynchrone, INPUT, false, DocString{"The asynchrone option controls the execution style: when true, it allows asynchronous operations with overlapping computation and communication, improving parallel performance. When false, it runs synchronously, ensuring sequential execution of operations and data updates."});

      inline std::string documentation() const override final
      {
	return R"EOF(  The StreamingLBM class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the streaming steps.)EOF";
      }


      inline void execute () override final
      {
	auto& data = *GridDataQ;
	auto& domain = *DomainQ;
	auto& traversals = *Traversals;
	grid<3>& Grid = domain.m_grid;

	// define functors
	streaming_step1<Q> step1 = {};
	streaming_step2<Q> step2 = {Grid};

	// get fields
	WrapperF<Q> pf = data.distributions();
	auto [pex, pey, pez] = data.exyz();

	if( *asynchrone )
	{
	  /*
	     constexpr Traversal Inside = Traversal::Inside;
	     constexpr Traversal Rest = Traversal::Ghost_Edge;

	     domain.m_ghost_manager.resize_request();
	     domain.m_ghost_manager.do_recv();
	     domain.m_ghost_manager.do_pack_send(pf, Grid.bx);

	     auto [ptr, size] = traversals.get_data<Inside>();
	     box<3> inside = Grid.build_box<Area::Local, Inside>();

	     parallel_for_id(ptr, size, step1, parallel_execution_context(), pf);
	     parallel_for_box(inside, step2, pf, pex, pey, pez);

	     domain.m_ghost_manager.wait_all();
	     domain.m_ghost_manager.do_unpack(pf, Grid.bx);

	     auto [ptr2, size2] = traversals.get_data<Rest>();

	     parallel_for_id(ptr2, size2, step1, parallel_execution_context(), pf);
	     parallel_for_ghost_edge(Grid, step2, pf, pex, pey, pez);
	   */
	}
	else
	{
	  // get traversal
	  auto [ptr, size] = traversals.get_data<Traversal::Real>();
	  // run kernel
	  parallel_for_id(ptr, size, step1, parallel_execution_context(), pf);
	  update_ghost(domain, pf);
	  box<3> extend = Grid.build_box<Area::Local, Traversal::Extend>();
#ifdef ONIKA_CUDA_VERSION
	  cuda_parallel_for_box(extend, step2, pf, pex, pey, pez);
#else
	  parallel_for_box(extend, step2, pf, pex, pey, pez);
#endif
	}
      }
  };

  using StreamingLBM3D19Q = StreamingLBM<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "streaming", make_compatible_operator<StreamingLBM3D19Q>);
  }
}

