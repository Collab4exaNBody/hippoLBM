#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <hippoLBM/grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/collision/streaming.hpp>
#include <grid/update_ghost.hpp>
#include <grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class StreamingLBM : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( LBMDomain<Q>, lbm_domain, INPUT, REQUIRED);
      ADD_SLOT( bool, asynchrone, INPUT, false, DocString{"The asynchrone option controls the execution style: when true, it allows asynchronous operations with overlapping computation and communication, improving parallel performance. When false, it runs synchronously, ensuring sequential execution of operations and data updates."});

      inline std::string documentation() const override final
      {
        return R"EOF(  The StreamingLBM class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the streaming steps.)EOF";
      }


      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& domain = *lbm_domain;
        auto& traversals = *Traversals;
        grid<3>& Grid = domain.m_grid;
        auto [ptr, size] = traversals.get_levels();

        // get fields
        FieldView<Q> pf = data.distributions();
        auto [pex, pey, pez] = data.exyz();

        // define functors
        streaming_step1<Q, Traversal::Real> step1 = {ptr, pf};
        streaming_step2<Q, Traversal::Extend> step2 = {ptr, Grid, pf, pex, pey, pez};

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        if( *asynchrone )
        {
          /*
             constexpr Traversal Inside = Traversal::Inside;
             constexpr Traversal Rest = Traversal::Ghost_Edge;

             domain.m_ghost_manager.resize_request();
             domain.m_ghost_manager.do_recv();
             domain.m_ghost_manager.do_pack_send(pf, Grid.bx);

             auto [ptr, size] = traversals.get_data<Inside>();
             box<3> inside = Grid.build_box<Area::Local, Inside>();

             parallel_for_id(ptr, size, step1, parallel_execution_context(), pf);
             parallel_for_box(inside, step2, pf, pex, pey, pez);

             domain.m_ghost_manager.wait_all();
             domain.m_ghost_manager.do_unpack(pf, Grid.bx);

             auto [ptr2, size2] = traversals.get_data<Rest>();

             parallel_for_id(ptr2, size2, step1, parallel_execution_context(), pf);
             parallel_for_ghost_edge(Grid, step2, pf, pex, pey, pez);
           */
        }
        else
        {
          // run kernel
          parallel_for_simple(size, step1, parallel_execution_context("streaming_step1"));
          update_ghost(domain, pf, par_exec_ctx);
          parallel_for_simple(size, step2, parallel_execution_context("streaming_step2"));
/*
          box<3> extend = Grid.build_box<Area::Local, Traversal::Extend>();
          onika::parallel::ParallelExecutionSpace<3> parallel_range = set(extend);        
          parallel_for(parallel_range, step2, parallel_execution_context("streaming_step2"));
*/
        }
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(streaming)
  {
    OperatorNodeFactory::instance()->register_factory( "streaming", make_variant_operator<StreamingLBM>);
  }
}

