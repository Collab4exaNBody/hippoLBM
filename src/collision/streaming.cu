#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid/domain_lbm.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/collision/streaming.hpp>
#include <grid/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class StreamingLBM : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);
      ADD_SLOT( bool, asynchrone, INPUT, false, DocString{"The asynchrone option controls the execution style: when true, it allows asynchronous operations with overlapping computation and communication, improving parallel performance. When false, it runs synchronously, ensuring sequential execution of operations and data updates."});

      inline std::string documentation() const override final
      {
        return R"EOF(  The StreamingLBM class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the streaming steps.)EOF";
      }


      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& domain = *DomainQ;
        auto& traversals = *Traversals;
        grid<3>& Grid = domain.m_grid;

        // get fields
        FieldView<Q> pf = data.distributions();
        auto [pex, pey, pez] = data.exyz();

        // define functors
        streaming_step1<Q> step1 = {};
        streaming_step2<Q> step2 = {Grid, pf, pex, pey, pez};

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        if( *asynchrone )
        {
          /*
             constexpr Traversal Inside = Traversal::Inside;
             constexpr Traversal Rest = Traversal::Ghost_Edge;

             domain.m_ghost_manager.resize_request();
             domain.m_ghost_manager.do_recv();
             domain.m_ghost_manager.do_pack_send(pf, Grid.bx);

             auto [ptr, size] = traversals.get_data<Inside>();
             box<3> inside = Grid.build_box<Area::Local, Inside>();

             parallel_for_id(ptr, size, step1, parallel_execution_context(), pf);
             parallel_for_box(inside, step2, pf, pex, pey, pez);

             domain.m_ghost_manager.wait_all();
             domain.m_ghost_manager.do_unpack(pf, Grid.bx);

             auto [ptr2, size2] = traversals.get_data<Rest>();

             parallel_for_id(ptr2, size2, step1, parallel_execution_context(), pf);
             parallel_for_ghost_edge(Grid, step2, pf, pex, pey, pez);
           */
        }
        else
        {
          // get traversal
          auto [ptr, size] = traversals.get_data<Traversal::Real>();

          // run kernel
          parallel_for_id(ptr, size, step1, parallel_execution_context("streaming_step1"), pf);
          update_ghost(domain, pf, par_exec_ctx);
          box<3> extend = Grid.build_box<Area::Local, Traversal::Extend>();
          onika::parallel::ParallelExecutionSpace<3> parallel_range = set(extend);        
          parallel_for(parallel_range, step2, parallel_execution_context("streaming_step2"));
        }
      }
  };

  using StreamingLBM3D19Q = StreamingLBM<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "streaming", make_compatible_operator<StreamingLBM3D19Q>);
  }
}

