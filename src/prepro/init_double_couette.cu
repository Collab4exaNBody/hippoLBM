#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/lbm_parameters.hpp>
#include <hippoLBM/grid/parallel_for_core.cu>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/prepro/double_couette.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class InitDoubleCouette : public OperatorNode
  {
    ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);
    ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( Vec3d, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = 0), enforcing the Neumann condition."});
    ADD_SLOT( LBMParameters, Params, INPUT, REQUIRED, DocString{"Contains global LBM simulation parameters"});
    ADD_SLOT( std::string, dimension, INPUT, REQUIRED, DocString{"Choose the dimension."});
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( 
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *fields;
      auto& params = *Params;
      int3d domain_size = domain->size();
      LBMGrid& grid = domain->grid();

      // define variables
      Vec3d Uc = (*U) / params.celerity;
      
      // get fields 
      FieldView<Q> pf = data.distributions();
      auto [pex, pey, pez] = data.exyz();
      const double * const pw = data.weights();

      // get traversal
      Box3D real = grid.build_box<Area::Local, Traversal::Real>();
      onika::parallel::ParallelExecutionSpace<3> parallel_range = set(real);

			if( *dimension == "X")
			{
        // define variables
        Vec3d dU = Uc / (0.5 * (domain_size[DIMX] - 1));
				// define functors
				InitDoubleCouetteFunc<Q,DIMX> func = { grid, pf, dU, Uc, pex, pey, pez, pw };
				// run kernel
				parallel_for(parallel_range, func, parallel_execution_context("init_double_couette_dim_x"));
			}
      else if(*dimension == "Y")
			{
        // define variables
        Vec3d dU = Uc / (0.5 * (domain_size[DIMY] - 1));
				// define functors
				InitDoubleCouetteFunc<Q,DIMY> func = { grid, pf, dU, Uc, pex, pey, pez, pw };
				// run kernel
				parallel_for(parallel_range, func, parallel_execution_context("init_double_couette_dim_y"));
			}
      else if(*dimension == "Z")
			{
        lout << "Prepro double couette starting ... dim Z" << std::endl;
        // define variables
        Vec3d dU = Uc / (0.5 * (domain_size[DIMZ] - 1));
        lout << "Uc: [" << Uc << "]" << std::endl;
        lout << "dU: [" << dU << "]" << std::endl;
				// define functors
				InitDoubleCouetteFunc<Q,DIMZ> func = { grid, pf, dU, Uc, pex, pey, pez, pw };
				// run kernel
				parallel_for(parallel_range, func, parallel_execution_context("init_double_couette_dim_z"));
        lout << "Prepro double couette ending ... dim Z " << std::endl;
			}
      else
      {
        lout << "[init_double_couette] Please, select a valide dimension \"X\", \"Y\", or \"Z\"." << std::endl; 
        std::exit(EXIT_FAILURE);
      }
		}
	};

	// === register factories ===  
	ONIKA_AUTORUN_INIT(init_double_couette)
	{
		OperatorNodeFactory::instance()->register_factory( "init_double_couette", make_variant_operator<InitDoubleCouette>);
	}
}
