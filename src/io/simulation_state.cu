#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/grid/make_variant_operator.hpp>
#include <hippoLBM/compute/reduce.hpp>
#include <hippoLBM/io/simulation_state.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;
  using namespace onika::memory;


  template<int Q>
    class SimulationState : public OperatorNode
  {
    public:

      ADD_SLOT(MPI_Comm, mpi, INPUT, MPI_COMM_WORLD);
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
      ADD_SLOT( SimulationStatistics, simulation_statistics, OUTPUT, DocString{"Contains general information about the LBM grid, such as minimum and maximum fluid velocity."});
      ADD_SLOT( CudaMMVector<SimulationStatistics>, scratch, PRIVATE);

      inline std::string documentation() const override final
      {
        return R"EOF( 
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *fields;
        auto& region = *grid_region;

        auto& buffer = *scratch;
        reset_scratch(buffer, parallel_execution_context("reset_scratch"));
        //buffer[0].display();

        // get fields
        double * const pm0 = data.densities();
        FieldView<3> pm1 = data.flux();

        ComuteSimulationStateFunc func = {pm0, pm1};

        local_reduce(func, buffer, parallel_execution_context("comute_simulation_state"), region);
        local_reduce_sync();
        // reduce on master
        SimulationStatistics local = buffer[0];
        SimulationStatistics global = {};
        int master = 0;
        MPI_Reduce(&local.sum_density, &global.sum_density, 1, MPI_DOUBLE, MPI_SUM, master, *mpi);
        MPI_Reduce(&local.min_velocity_norm, &global.min_velocity_norm, 1, MPI_DOUBLE, MPI_MIN, master, *mpi);
        MPI_Reduce(&local.max_velocity_norm, &global.max_velocity_norm, 1, MPI_DOUBLE, MPI_MAX, master, *mpi);
               
        *simulation_statistics = global;
     }
  };

  using SimulationState3D19Q = SimulationState<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(SimulationState)
  {
    OperatorNodeFactory::instance()->register_factory( "hippolbm_simulation_state", make_variant_operator<SimulationState>);
  }
}

