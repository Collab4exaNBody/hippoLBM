#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <grid/make_variant_operator.hpp>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/init_obst.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class InitObstLBM : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT);

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& domain = *LBMDomain;
        init_obst func = {onika::cuda::vector_data(data.obst)};
        constexpr Area A = Area::Local;
        constexpr Traversal Tr = Traversal::All;
        parallel_for_id<A,Tr>(domain.m_grid, func, parallel_execution_context());       
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_obstacles)
  {
    OperatorNodeFactory::instance()->register_factory( "init_obst", make_variant_operator<InitObstLBM>);
  }
}

