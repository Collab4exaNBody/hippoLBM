#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <grid/domain_lbm.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/init_obst.hpp>

namespace hippoLBM
{
	using namespace onika;
	using namespace scg;

	template<int Q>
		class InitObstLBM : public OperatorNode
	{
		public:
			ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);
			ADD_SLOT( lbm_fields<Q>, GridDataQ, INPUT_OUTPUT);

			inline void execute () override final
			{
        auto& data = *GridDataQ;
        auto& domain = *DomainQ;
				init_obst func = {onika::cuda::vector_data(data.obst)};
				constexpr Area A = Area::Local;
				constexpr Traversal Tr = Traversal::All;
		    parallel_for_id<A,Tr>(domain.m_grid, func, parallel_execution_context());       
			}
	};

	using InitObstLBM3D19Q = InitObstLBM<19>;

	// === register factories ===  
	ONIKA_AUTORUN_INIT(parallel_for_benchmark)
	{
		OperatorNodeFactory::instance()->register_factory( "init_obst", make_compatible_operator<InitObstLBM3D19Q>);
	}
}

