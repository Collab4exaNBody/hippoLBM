#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <hippoLBM/grid/make_variant_operator.hpp>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/parallel_for_core.cu>
#include <hippoLBM/grid/init_obst.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class InitObstLBM : public OperatorNode
  {
    public:
      ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED);

      inline void execute () override final
      {
        auto& data = *fields;
        init_obst func = {onika::cuda::vector_data(data.obst)};
        constexpr Area A = Area::Local;
        constexpr Traversal Tr = Traversal::All;
        parallel_for_id<A,Tr>(domain->m_grid, func, parallel_execution_context());       
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_obstacles)
  {
    OperatorNodeFactory::instance()->register_factory( "init_obst", make_variant_operator<InitObstLBM>);
  }
}

