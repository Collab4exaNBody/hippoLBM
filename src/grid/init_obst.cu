#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <hippoLBM/grid/make_variant_operator.hpp>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/parallel_for_core.cu>
#include <hippoLBM/grid/init_obst.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class InitObstLBM : public OperatorNode
  {
    public:
      ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT);

      inline void execute () override final
      {
        auto& data = *fields;
        init_obst func = {onika::cuda::vector_data(data.obst)};
        constexpr Area A = Area::Local;
        constexpr Traversal Tr = Traversal::All;
        parallel_for_id<A,Tr>(domain->m_grid, func, parallel_execution_context());       
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_obstacles)
  {
    OperatorNodeFactory::instance()->register_factory( "init_obst", make_variant_operator<InitObstLBM>);
  }
}

