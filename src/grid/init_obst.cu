#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <grid/make_variant_operator.hpp>
#include <hippoLBM/grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/init_obst.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class InitObstLBM : public OperatorNode
  {
    public:
      ADD_SLOT( LBMDomain<Q>, lbm_domain, INPUT, REQUIRED);
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT);

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& domain = *lbm_domain;
        init_obst func = {onika::cuda::vector_data(data.obst)};
        constexpr Area A = Area::Local;
        constexpr Traversal Tr = Traversal::All;
        parallel_for_id<A,Tr>(domain.m_grid, func, parallel_execution_context());       
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_obstacles)
  {
    OperatorNodeFactory::instance()->register_factory( "init_obst", make_variant_operator<InitObstLBM>);
  }
}

