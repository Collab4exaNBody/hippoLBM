#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/lbm_domain.hpp>
#include <grid/update_ghost.hpp>
#include <grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class UpdateGhost : public OperatorNode
  {
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);

    public:

      inline std::string documentation() const override final
      {
        return R"EOF(  A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& domain = *LBMDomain;

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        // get fields
        FieldView<Q> pf = data.distributions();
        update_ghost(domain, pf, par_exec_ctx);
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(update_ghost)
  {
    OperatorNodeFactory::instance()->register_factory( "update_ghost", make_variant_operator<UpdateGhost>);
  }
}

