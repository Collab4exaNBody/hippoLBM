#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/lbm_domain.hpp>
#include <grid/update_ghost.hpp>
#include <grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class UpdateGhost : public OperatorNode
  {
    ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);

    public:

    inline std::string documentation() const override final
    {
      return R"EOF(  A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *LBMFieds;
      auto& domain = *LBMDomain;

      // capture the parallel execution context
      auto par_exec_ctx = [this] (const char* exec_name)
      { 
        return this->parallel_execution_context(exec_name);
      };

      // get fields
      FieldView<Q> pf = data.distributions();
      update_ghost(domain, pf, par_exec_ctx);
    }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(update_ghost)
  {
    OperatorNodeFactory::instance()->register_factory( "update_ghost", make_variant_operator<UpdateGhost>);
  }
}

