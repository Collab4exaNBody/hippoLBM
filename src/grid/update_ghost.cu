#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/domain.hpp>
#include <grid/update_ghost.hpp>
#include <hippoLBM/grid/make_variant_operator.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class UpdateGhost : public OperatorNode
  {
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);

    public:

      inline std::string documentation() const override final
      {
        return R"EOF(  A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *fields;

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        // get fields
        FieldView<Q> pf = data.distributions();
        update_ghost(*domain, pf, par_exec_ctx);
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(update_ghost)
  {
    OperatorNodeFactory::instance()->register_factory( "update_ghost", make_variant_operator<UpdateGhost>);
  }
}

