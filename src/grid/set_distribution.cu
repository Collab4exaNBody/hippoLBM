#include "hip/hip_runtime.h"
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/grid/set_distribution.hpp>
#include <grid/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class SetDistributionsLBM : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT);
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED);
      ADD_SLOT( AABB, bounds, INPUT, OPTIONAL, DocString{"Domain's bounds"});
      ADD_SLOT( double, value, INPUT, double(1) );
      ADD_SLOT( bool, do_update, INPUT, false);

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& traversals = *Traversals;
        lbm_domain<Q>& domain = *LBMDomain;

        FieldView pf = data.distributions();
        const double * const pw = data.weights();

        // define kernel
        init_distributions<Q> func = {*value};

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        if(bounds.has_value())
        {
          grid<3>& Grid = domain.m_grid;

          auto& bound = *bounds;
          Vec3d min = bound.bmin;
          Vec3d max = bound.bmax;
          double Dx = Grid.dx;
          point<3> _min = {int(min.x/Dx), int(min.y/Dx), int(min.z/Dx)};
          point<3> _max = {int(max.x/Dx), int(max.y/Dx), int(max.z/Dx)};

          box<3> global_wall_box = {_min, _max};
          global_wall_box.print();

          auto [is_inside_subdomain, wall_box] = Grid.restrict_box_to_grid<Area::Local, Traversal::Extend>(global_wall_box);
          wall_box.print();
          if( !is_inside_subdomain ) return;

          for(int z = wall_box.start(2) ; z <= wall_box.end(2) ; z++)
            for(int y = wall_box.start(1) ; y <= wall_box.end(1) ; y++)
              for(int x = wall_box.start(0) ; x <= wall_box.end(0) ; x++)
              {
                const int idx = Grid(x,y,z);
                func(idx, pf, pw);
              }

        }
        else  // all domain
        { 
          if( *do_update )
          {
            auto [ptr, size] = traversals.get_data<Traversal::Real>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
            update_ghost(domain, pf, par_exec_ctx);
          }
          else
          {
            auto [ptr, size] = traversals.get_data<Traversal::All>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
          }
        }
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_distributions)
  {
    OperatorNodeFactory::instance()->register_factory( "set_distribution", make_variant_operator<SetDistributionsLBM>);
  }
}
