#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/parallel_for_core.cu>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/grid/set_distribution.hpp>
#include <hippoLBM/grid/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class SetDistributionsLBM : public OperatorNode
  {
    public:
      ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT);
      ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED);
      ADD_SLOT( AABB, bounds, INPUT, OPTIONAL, DocString{"Domain's bounds"});
      ADD_SLOT( double, value, INPUT, double(1) );
      ADD_SLOT( bool, do_update, INPUT, false);

      inline void execute () override final
      {
        auto& data = *fields;
        auto& traversals = *grid_region;
        LBMDomain<Q>& Domain = *domain;
        LBMGrid& Grid = Domain.m_grid;
        GridIJKtoIdx ijk_to_idx(Grid);

        FieldView pf = data.distributions();
        const double * const pw = data.weights();

        // define kernel
        init_distributions<Q> func = {*value, ijk_to_idx};

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        if(bounds.has_value())
        {

          auto& bound = *bounds;
          Vec3d min = bound.bmin;
          Vec3d max = bound.bmax;
          double Dx = Grid.dx;
          Point3D _min = {int(min.x/Dx), int(min.y/Dx), int(min.z/Dx)};
          Point3D _max = {int(max.x/Dx), int(max.y/Dx), int(max.z/Dx)};

          Box3D global_wall_box = {_min, _max};

          auto [is_inside_subdomain, wall_box] = Grid.restrict_box_to_grid<Area::Local, Traversal::Extend>(global_wall_box);
          if( !is_inside_subdomain ) return;

          parallel_for(wall_box, func, parallel_execution_context(), pf, pw);

        }
        else  // all domain
        { 
          if( *do_update )
          {
            auto [ptr, size] = traversals.get_data<Traversal::Real>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
            update_ghost(Domain, pf, par_exec_ctx);
          }
          else
          {
            auto [ptr, size] = traversals.get_data<Traversal::All>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
          }
        }
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_distributions)
  {
    OperatorNodeFactory::instance()->register_factory( "set_distribution", make_variant_operator<SetDistributionsLBM>);
  }
}
