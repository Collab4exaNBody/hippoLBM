#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <grid/init_distributions.hpp>
#include <grid/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class InitDistributionsLBM : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT);
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED);
      ADD_SLOT( AABB, bounds, INPUT, OPTIONAL, DocString{"Domain's bounds"});
      ADD_SLOT( double, tmp_coeff, INPUT, double(1) );
      ADD_SLOT( bool, do_update, INPUT, false);

      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& traversals = *Traversals;
        lbm_domain<Q>& domain = *LBMDomain;

        FieldView pf = data.distributions();
        const double * const pw = data.weights();

        // define kernel
        init_distributions<Q> func = {*tmp_coeff};

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        if(bounds.has_value())
        {
          grid<3>& Grid = domain.m_grid;

          auto& bound = *bounds;
          Vec3d min = bound.bmin;
          Vec3d max = bound.bmax;
          double Dx = Grid.dx;
          point<3> _min = {int(min.x/Dx), int(min.y/Dx), int(min.z/Dx)};
          point<3> _max = {int(max.x/Dx), int(max.y/Dx), int(max.z/Dx)};

          box<3> global_wall_box = {_min, _max};
          global_wall_box.print();

          auto [is_inside_subdomain, wall_box] = Grid.restrict_box_to_grid<Area::Local, Traversal::Extend>(global_wall_box);
          wall_box.print();
          if( !is_inside_subdomain ) return;

          for(int z = wall_box.start(2) ; z <= wall_box.end(2) ; z++)
            for(int y = wall_box.start(1) ; y <= wall_box.end(1) ; y++)
              for(int x = wall_box.start(0) ; x <= wall_box.end(0) ; x++)
              {
                const int idx = Grid(x,y,z);
                func(idx, pf, pw);
              }

        }
        else  // all domain
        { 
          if( *do_update )
          {
            auto [ptr, size] = traversals.get_data<Traversal::Real>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
            update_ghost(domain, pf, par_exec_ctx);
          }
          else
          {
            auto [ptr, size] = traversals.get_data<Traversal::All>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
          }
        }
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(init_distributions)
  {
    OperatorNodeFactory::instance()->register_factory( "init_distributions", make_variant_operator<InitDistributionsLBM>);
  }
}
