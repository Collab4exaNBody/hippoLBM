#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <grid_lbm/init_distributions.hpp>
#include <grid_lbm/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;

  template<int Q>
    class InitDistributionsLBM : public OperatorNode
  {
    public:
      ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);
      ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT);
      ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED);
      ADD_SLOT( AABB, bounds, INPUT, OPTIONAL, DocString{"Domain's bounds"});
      ADD_SLOT( double, tmp_coeff, INPUT, double(1) );
      ADD_SLOT( bool, do_update, INPUT, false);

      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& traversals = *Traversals;
        domain_lbm<Q>& domain = *DomainQ;

        WrapperF pf = data.distributions();
        const double * const pw = data.weights();

        init_distributions<Q> func = {*tmp_coeff};

        if(bounds.has_value())
        {
          grid<3>& Grid = domain.m_grid;

          auto& bound = *bounds;
          Vec3d min = bound.bmin;
          Vec3d max = bound.bmax;
          double Dx = Grid.dx;
          point<3> _min = {int(min.x/Dx), int(min.y/Dx), int(min.z/Dx)};
          point<3> _max = {int(max.x/Dx), int(max.y/Dx), int(max.z/Dx)};

          box<3> global_wall_box = {_min, _max};
          global_wall_box.print();

          auto [is_inside_subdomain, wall_box] = Grid.restrict_box_to_grid<Area::Local, Traversal::Extend>(global_wall_box);
          wall_box.print();
          if( !is_inside_subdomain ) return;

          for(int z = wall_box.start(2) ; z <= wall_box.end(2) ; z++)
            for(int y = wall_box.start(1) ; y <= wall_box.end(1) ; y++)
              for(int x = wall_box.start(0) ; x <= wall_box.end(0) ; x++)
              {
                const int idx = Grid(x,y,z);
                func(idx, pf, pw);
              }

        }
        else  // all domain
        { 
          if( *do_update )
          {
            auto [ptr, size] = traversals.get_data<Traversal::Real>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
            update_ghost(domain, pf);
          }
          else
          {
            auto [ptr, size] = traversals.get_data<Traversal::All>();
            parallel_for_id(ptr, size, func, parallel_execution_context(), pf, pw);
          }
        }
      }
  };

  using InitDistributionsLBM3D19Q = InitDistributionsLBM<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(parallel_for_benchmark)
  {
    OperatorNodeFactory::instance()->register_factory( "init_distributions", make_compatible_operator<InitDistributionsLBM3D19Q>);
  }
}
