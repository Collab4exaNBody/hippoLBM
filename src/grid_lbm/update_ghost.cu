#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/update_ghost.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class UpdateGhost : public OperatorNode
  {
      ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);

    public:

      inline std::string documentation() const override final
      {
        return R"EOF(  A functor for computing macroscopic variables (densities and flux) for lattice Boltzmann method.
        )EOF";
      }

      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& domain = *DomainQ;

        // capture the parallel execution context
        auto par_exec_ctx = [this] (const char* exec_name)
        { 
          return this->parallel_execution_context(exec_name);
        };

        // get fields
        WrapperF<Q> pf = data.distributions();
        update_ghost(domain, pf, par_exec_ctx);
      }
  };

  using UpdateGhost3D19Q = UpdateGhost<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(update_ghost)
  {
    OperatorNodeFactory::instance()->register_factory( "update_ghost", make_compatible_operator<UpdateGhost3D19Q>);
  }
}

