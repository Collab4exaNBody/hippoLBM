#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <grid/enum.hpp>
#include <hippoLBM/grid/domain.hpp>
#include <grid/comm.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back_manager.hpp>
#include <hippoLBM/bcs/cavity.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

	template<int Dim, Side S, int Q>
		class Cavity : public OperatorNode
	{
		typedef std::array<double,3> readVec3;
		ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);
		ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
		ADD_SLOT( readVec3, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = lz), enforcing the Cavity condition."});
		ADD_SLOT( bounce_back_manager<Q>, bbmanager, INPUT_OUTPUT, REQUIRED);

		public:
		inline std::string documentation() const override final
		{
			return R"EOF( This operator enforces a Cavity boundary condition at z = lz in an LBM simulation. 
                      The Cavity boundary condition ensures that the gradient of the distribution function 
                      follows a prescribed value
        )EOF";
		}

		inline void execute () override final
		{
			auto& data = *fields;
			auto& bb = *bbmanager;
			auto [lx, ly, lz] = domain->domain_size;
			auto [ux,uy,uz] = *U;

			// define functors
			cavity<Dim, S, Q> bcs = {};

			// get fields
			constexpr int idx = helper_dim_idx<Dim,S>();
			FieldView<5> pfi = bb.get_data(idx);
			int * const pobst = data.obstacles();
			auto [pex, pey, pez] = data.exyz();
			const double * const pw = data.weights();

			// initialize coefficients
			bcs.compute_coeff(ux, uy, uz, pw, pex, pey, pez, lx, ly, lz);

			// run kernel
			auto params = make_tuple(pobst, pfi);
			parallel_for_id_runner runner = {bcs, params};
			parallel_for(pfi.num_elements, runner, parallel_execution_context(), ParallelForOptions());
		}
	};

	template<int Q> using CavityZ0_3D19Q = Cavity<DIMZ, Side::Left, Q>;
	template<int Q> using CavityZL_3D19Q = Cavity<DIMZ, Side::Right,Q>;

	// === register factories ===  
	ONIKA_AUTORUN_INIT(cavity)
	{
		OperatorNodeFactory::instance()->register_factory( "cavity_z_0", make_variant_operator<CavityZ0_3D19Q>);
		OperatorNodeFactory::instance()->register_factory( "cavity_z_l", make_variant_operator<CavityZL_3D19Q>);
	}
}

