#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <grid_lbm/lbm_parameters.hpp>
#include <hippoLBM/bcs/bounce_back_manager.hpp>
#include <hippoLBM/bcs/cavity.hpp>

namespace hippoLBM
{
	using namespace onika;
	using namespace scg;
	using namespace onika::cuda;

	template<int Q>
		class CavityZL : public OperatorNode
	{
		typedef std::array<double,3> readVec3;
		ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
		ADD_SLOT( readVec3, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = lz), enforcing the Cavity condition."});
    ADD_SLOT( bounce_back_manager<Q>, bbmanager, INPUT_OUTPUT, REQUIRED);

		public:
		inline std::string documentation() const override final
		{
			return R"EOF( This operator enforces a Cavity boundary condition at z = lz in an LBM simulation. 
                      The Cavity boundary condition ensures that the gradient of the distribution function 
                      follows a prescribed value
        )EOF";
		}

		inline void execute () override final
		{
			auto& data = *GridDataQ;
			auto& bb = *bbmanager;

			// define functors
			cavity_z_l<Q> bcs = {};

			auto [ux,uy,uz] = *U;

			// get fields
      constexpr int dimZ = 2;
      constexpr int idx = helper_dim_idx<dimZ,Direction::Right>();
      WrapperF<5> pfi = bb.get_data(idx);
			int * const pobst = data.obstacles();
      auto [pex, pey, pez] = data.exyz();
      const double * const pw = data.weights();

			// run kernel
      auto params = make_tuple(pobst, pfi, ux, uy, uz, pw, pex, pey, pez, 30,30,30);
      parallel_for_id_runner runner = {bcs, params};
      //onika::lout << " Fi size: "<< pfi.N << std::endl;
      parallel_for(pfi.N, runner, parallel_execution_context(), ParallelForOptions());
			//parallel_for_simple(size, bcs, parallel_execution_context(), pobst, pf, ux, uy, uz, pw, pex, pey, pez, 30,30,30);
		}
	};

	using CavityZL3D19Q = CavityZL<19>;

	// === register factories ===  
	ONIKA_AUTORUN_INIT()
	{
		OperatorNodeFactory::instance()->register_factory( "cavity_z_l", make_compatible_operator<CavityZL3D19Q>);
	}
}

