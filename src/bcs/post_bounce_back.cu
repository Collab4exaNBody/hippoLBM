#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>
#include <hippoLBM/bcs/bounce_back_manager.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class PostBounceBack : public OperatorNode
  {
    ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( bounce_back_manager<Q>, bbmanager, INPUT_OUTPUT);
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( 
        )EOF";
    }

    template<int dim, Side dir> 
      void launcher(traversal_lbm& traversals, FieldView<Q>& pf, bounce_back_manager<Q>& bbm)
      {
        constexpr int idx = helper_dim_idx<dim,dir>();
        FieldView<bounce_back_manager<Q>::Un> pfi = bbm.get_data(idx);
        if( pfi.num_elements> 0 )
        {
          constexpr Traversal Tr = get_traversal<dim, dir>();
          auto [ptr, size] = traversals.get_data<Tr>();

          assert(ptr != nullptr);
          assert(pfi.num_elements == int(size));

          ParallelForOptions opts;
          opts.omp_scheduling = OMP_SCHED_STATIC;
          post_bounce_back<dim, dir, Q> kernel = {ptr};
          auto params = make_tuple(pf, pfi);
          parallel_for_id_runner runner = {kernel, params}; //pf, pfi};
          parallel_for(size, runner, parallel_execution_context(), opts);
      }
  }

  inline void execute () override final
  {
    auto& data = *LBMFieds;
    auto& traversals = *Traversals;

    // storage
    auto& bb = *bbmanager;

    // define functors

    // get fields
    FieldView<Q> pf = data.distributions();

    // for clarity
    constexpr int dim_x = 0;
    constexpr int dim_y = 1;
    constexpr int dim_z = 2;
    launcher<dim_x, Side::Left>(traversals, pf, bb);
    launcher<dim_x, Side::Right>(traversals, pf, bb);
    launcher<dim_y, Side::Left>(traversals, pf, bb);
    launcher<dim_y, Side::Right>(traversals, pf, bb);
    launcher<dim_z, Side::Left>(traversals, pf, bb);
    launcher<dim_z, Side::Right>(traversals, pf, bb);
  }
};

// === register factories ===  
ONIKA_AUTORUN_INIT()
{
  OperatorNodeFactory::instance()->register_factory( "post_bounce_back", make_variant_operator<PostBounceBack>);
}
}

