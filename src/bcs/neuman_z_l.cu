#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <grid_lbm/lbm_parameters.hpp>
#include <hipoLBM/bcs/neumann.hpp>

namespace hipoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class NeumannZL : public OperatorNode
  {
    typedef std::array<double,3> readVec3;
    ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( readVec3, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = lz), enforcing the Neumann condition."});

    public:
    inline std::string documentation() const override final
    {
      return R"EOF( This operator enforces a Neumann boundary condition at z = lz in an LBM simulation. 
                      The Neumann boundary condition ensures that the gradient of the distribution function 
                      follows a prescribed value
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *GridDataQ;
      auto& traversals = *Traversals;

      // define functors
      neumann_z_l<Q> neumann = {};

      auto [ux,uy,uz] = *U;

      // get fields
      WrapperF<Q> pf = data.distributions();
      int * const pobst = data.obstacles();

      // get traversal
      auto [ptr, size] = traversals.get_data<Traversal::Plan_xy_l>();
      // run kernel
      parallel_for_id(ptr, size, neumann, parallel_execution_context(), pobst, pf, ux, uy, uz);
    }
  };

  using NeumannZL3D19Q = NeumannZL<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "neumann_z_l", make_compatible_operator<NeumannZL3D19Q>);
  }
}

