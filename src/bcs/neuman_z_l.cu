#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/compute/parallel_for_core.hpp>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/grid/lbm_parameters.hpp>
#include <hippoLBM/bcs/neumann.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class NeumannZL : public OperatorNode
  {
    typedef std::array<double,3> readVec3;
    ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( readVec3, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = lz), enforcing the Neumann condition."});

    public:
    inline std::string documentation() const override final
    {
      return R"EOF( This operator enforces a Neumann boundary condition at z = lz in an LBM simulation. 
                      The Neumann boundary condition ensures that the gradient of the distribution function 
                      follows a prescribed value
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *fields;
      auto& traversals = *grid_region;

      // define functors
      neumann_z_l<Q> neumann = {};

      auto [ux,uy,uz] = *U;

      // get fields
      FieldView<Q> pf = data.distributions();
      int * const pobst = data.obstacles();

      // get traversal
      auto [ptr, size] = traversals.get_data<Traversal::Plan_xy_l>();
      if( size == 0) return;
      // run kernel
      parallel_for_id(ptr, size, neumann, parallel_execution_context(), pobst, pf, ux, uy, uz);
    }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(neumann_z_l)
  {
    OperatorNodeFactory::instance()->register_factory( "neumann_z_l", make_variant_operator<NeumannZL>);
  }
}

