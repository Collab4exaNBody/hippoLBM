#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class WallBounceBack : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( LBMDomain<Q>, lbm_domain, INPUT, REQUIRED);

      inline std::string documentation() const override final
      {
        return R"EOF(  The WallBounceBack class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the wall bounce back steps.)EOF";
      }


      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& domain = *lbm_domain;
        grid<3>& Grid = domain.m_grid;

        // get fields
        const int* const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        auto [pex, pey, pez] = data.exyz();

        // define functors
        wall_bounce_back<Q> func = {Grid, pobst, pf, pex, pey, pez};

        // run kernel
        box<3> extend = Grid.build_box<Area::Local, Traversal::Extend>();
        onika::parallel::ParallelExecutionSpace<3> parallel_range = set(extend);
        parallel_for(parallel_range, func, parallel_execution_context("wall_bounce_back"));
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(wall_bounce_back)
  {
    OperatorNodeFactory::instance()->register_factory( "wall_bounce_back", make_variant_operator<WallBounceBack>);
  }
}

