#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */


#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class WallBounceBack : public OperatorNode
  {
    public:
      ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( lbm_domain<Q>, LBMDomain, INPUT, REQUIRED);

      inline std::string documentation() const override final
      {
        return R"EOF(  The WallBounceBack class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the wall bounce back steps.)EOF";
      }


      inline void execute () override final
      {
        auto& data = *LBMFieds;
        auto& domain = *LBMDomain;
        grid<3>& Grid = domain.m_grid;

        // get fields
        const int* const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        auto [pex, pey, pez] = data.exyz();

        // define functors
        wall_bounce_back<Q> func = {Grid, pobst, pf, pex, pey, pez};

        // run kernel
        box<3> extend = Grid.build_box<Area::Local, Traversal::Extend>();
        onika::parallel::ParallelExecutionSpace<3> parallel_range = set(extend);
        parallel_for(parallel_range, func, parallel_execution_context("wall_bounce_back"));
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(wall_bounce_back)
  {
    OperatorNodeFactory::instance()->register_factory( "wall_bounce_back", make_variant_operator<WallBounceBack>);
  }
}

