#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */


#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class WallBounceBack : public OperatorNode
  {
    public:
      ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);

      inline std::string documentation() const override final
      {
        return R"EOF(  The WallBounceBack class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the wall bounce back steps.)EOF";
      }


      inline void execute () override final
      {
        auto& data = *fields;
        LBMGrid& Grid = domain->m_grid;

        // get fields
        const int* const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        auto [pex, pey, pez] = data.exyz();

        // define functors
        wall_bounce_back<Q> func = {Grid, pobst, pf, pex, pey, pez};

        // run kernel
        Box3D extend = Grid.build_box<Area::Local, Traversal::Extend>();
        onika::parallel::ParallelExecutionSpace<3> parallel_range = set(extend);
        parallel_for(parallel_range, func, parallel_execution_context("wall_bounce_back"));
      }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(wall_bounce_back)
  {
    OperatorNodeFactory::instance()->register_factory( "wall_bounce_back", make_variant_operator<WallBounceBack>);
  }
}

