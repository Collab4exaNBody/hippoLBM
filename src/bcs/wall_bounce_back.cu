#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types_yaml.h>
#include <onika/math/basic_types_stream.h>
#include <onika/math/basic_types_operators.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class WallBounceBack : public OperatorNode
  {
    public:
      ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
      ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);

      inline std::string documentation() const override final
      {
        return R"EOF(  The WallBounceBack class is described as part of the Lattice Boltzmann Method (LBM) implementation, specifically the wall bounce back steps.)EOF";
      }


      inline void execute () override final
      {
        auto& data = *GridDataQ;
        auto& domain = *DomainQ;
        grid<3>& Grid = domain.m_grid;

        // get fields
        const int* const pobst = data.obstacles();
        FieldView<Q> pf = data.distributions();
        auto [pex, pey, pez] = data.exyz();

        // define functors
        wall_bounce_back<Q> func = {Grid, pobst, pf, pex, pey, pez};

        // run kernel
        box<3> extend = Grid.build_box<Area::Local, Traversal::Extend>();
        onika::parallel::ParallelExecutionSpace<3> parallel_range = set(extend);
        parallel_for(parallel_range, func, parallel_execution_context("wall_bounce_back"));
      }
  };

  using WallBounceBack3D19Q = WallBounceBack<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "wall_bounce_back", make_compatible_operator<WallBounceBack3D19Q>);
  }
}

