#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/domain.hpp>
#include <hippoLBM/grid/comm.hpp>
#include <hippoLBM/grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <hippoLBM/compute/parallel_for_core.hpp>
#include <hippoLBM/grid/grid_region.hpp>
#include <hippoLBM/bcs/neumann.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class NeumannZ0 : public OperatorNode
  {
    typedef std::array<double,3> readVec3;
    ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( LBMGridRegion, grid_region, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( readVec3, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = 0), enforcing the Neumann condition."});
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( This operator enforces a Neumann boundary condition at z = 0 in an LBM simulation. 
                      The Neumann boundary condition ensures that the gradient of the distribution function 
                      follows a prescribed value.
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *fields;
      auto& traversals = *grid_region;

      // define functors
      neumann_z_0<Q> neumann = {};

      auto [ux,uy,uz] = *U;

      // get fields
      FieldView<Q> pf = data.distributions();
      int * const pobst = data.obstacles();

      // get traversal
      auto [ptr, size] = traversals.get_data<Traversal::Plan_xy_0>();
      // run kernel
      parallel_for_id(ptr, size, neumann, parallel_execution_context(), pobst, pf, ux, uy, uz);
    }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "neumann_z_0", make_variant_operator<NeumannZ0>);
  }
}
