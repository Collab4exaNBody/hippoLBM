#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/lbm_domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <grid/lbm_fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/bcs/neumann.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;

  template<int Q>
    class NeumannZ0 : public OperatorNode
  {
    typedef std::array<double,3> readVec3;
    ADD_SLOT( lbm_fields<Q>, LBMFieds, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( readVec3, U, INPUT, REQUIRED, DocString{"Prescribed velocity at the boundary (z = 0), enforcing the Neumann condition."});
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( This operator enforces a Neumann boundary condition at z = 0 in an LBM simulation. 
                      The Neumann boundary condition ensures that the gradient of the distribution function 
                      follows a prescribed value.
        )EOF";
    }

    inline void execute () override final
    {
      auto& data = *LBMFieds;
      auto& traversals = *Traversals;

      // define functors
      neumann_z_0<Q> neumann = {};

      auto [ux,uy,uz] = *U;

      // get fields
      FieldView<Q> pf = data.distributions();
      int * const pobst = data.obstacles();

      // get traversal
      auto [ptr, size] = traversals.get_data<Traversal::Plan_xy_0>();
      // run kernel
      parallel_for_id(ptr, size, neumann, parallel_execution_context(), pobst, pf, ux, uy, uz);
    }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "neumann_z_0", make_variant_operator<NeumannZ0>);
  }
}

