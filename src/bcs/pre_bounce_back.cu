#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>
#include <hippoLBM/bcs/bounce_back_manager.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;
  using BoolVector = std::vector<bool>;

  template<int Q>
    class PreBounceBack : public OperatorNode
  {
    ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);
    ADD_SLOT( bounce_back_manager<Q>, bbmanager, INPUT_OUTPUT);
    ADD_SLOT( BoolVector, periodic   , INPUT , REQUIRED );
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( 
        )EOF";
    }

    template<int dim, Side dir> 
      void launcher(traversal_lbm& traversals, FieldView<Q>& pf, bounce_back_manager<Q>& bbm)
      {
        int idx = helper_dim_idx<dim,dir>();
        FieldView<bounce_back_manager<Q>::Un> pfi = bbm.get_data(idx);
        if( pfi.num_elements > 0 )
        {
          constexpr Traversal Tr = get_traversal<dim, dir>();
          auto [ptr, size] = traversals.get_data<Tr>();
          assert(size == size_t(pfi.num_elements));
          assert(ptr != nullptr);

          ParallelForOptions opts;
          opts.omp_scheduling = OMP_SCHED_STATIC;
          pre_bounce_back<dim, dir, Q> kernel = {ptr};
          auto params = make_tuple(pf, pfi);
          parallel_for_id_runner runner = {kernel, params};
          parallel_for(size, runner, parallel_execution_context(), opts);
        }
      }

    inline void execute () override final
    {
      auto& data = *GridDataQ;
      auto& traversals = *Traversals;
      auto& domain = *DomainQ;
      grid<3>& Grid = domain.m_grid;

      // fill grid size;
      constexpr Area L = Area::Local;
      constexpr Traversal R = Traversal::All;
      //constexpr Traversal R = Traversal::Real;
      auto br = Grid.build_box<L, R>();
      onika::math::IJK local_grid_size(br.get_length(0), br.get_length(1), br.get_length(2));

      // storage
      auto& bb = *bbmanager;
      bb.resize_data(*periodic, local_grid_size, domain.MPI_coord, domain.MPI_grid_size);

      // get fields
      FieldView<Q> pf = data.distributions();

      // for clarity
      constexpr int dim_x = 0;
      constexpr int dim_y = 1;
      constexpr int dim_z = 2;
      launcher<dim_x, Side::Left>(traversals, pf, bb);
      launcher<dim_x, Side::Right>(traversals, pf, bb);
      launcher<dim_y, Side::Left>(traversals, pf, bb);
      launcher<dim_y, Side::Right>(traversals, pf, bb);
      launcher<dim_z, Side::Left>(traversals, pf, bb);
      launcher<dim_z, Side::Right>(traversals, pf, bb);
    }
  };

  using PreBounceBack3D19Q = PreBounceBack<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pre_bounce_back)
  {
    OperatorNodeFactory::instance()->register_factory( "pre_bounce_back", make_compatible_operator<PreBounceBack3D19Q>);
  }
}
