#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <hippoLBM/grid/make_variant_operator.hpp>
#include <onika/math/basic_types.h>
#include <hippoLBM/grid/domain.hpp>
#include <grid/comm.hpp>
#include <grid/enum.hpp>
#include <hippoLBM/grid/fields.hpp>
#include <grid/parallel_for_core.cu>
#include <grid/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>
#include <hippoLBM/bcs/bounce_back_manager.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;
  using BoolVector = std::vector<bool>;

  template<int Q>
    class PreBounceBack : public OperatorNode
  {
    ADD_SLOT( LBMFields<Q>, fields, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( LBMDomain<Q>, domain, INPUT, REQUIRED);
    ADD_SLOT( bounce_back_manager<Q>, bbmanager, INPUT_OUTPUT);
    ADD_SLOT( BoolVector, periodic   , INPUT , REQUIRED );
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( 
        )EOF";
    }

    template<int dim, Side dir> 
      void launcher(traversal_lbm& traversals, FieldView<Q>& pf, bounce_back_manager<Q>& bbm)
      {
        int idx = helper_dim_idx<dim,dir>();
        FieldView<bounce_back_manager<Q>::Un> pfi = bbm.get_data(idx);
        if( pfi.num_elements > 0 )
        {
          constexpr Traversal Tr = get_traversal<dim, dir>();
          auto [ptr, size] = traversals.get_data<Tr>();
          assert(size == size_t(pfi.num_elements));
          assert(ptr != nullptr);

          ParallelForOptions opts;
          opts.omp_scheduling = OMP_SCHED_STATIC;
          pre_bounce_back<dim, dir, Q> kernel = {ptr};
          auto params = make_tuple(pf, pfi);
          parallel_for_id_runner runner = {kernel, params};
          parallel_for(size, runner, parallel_execution_context(), opts);
        }
      }

    inline void execute () override final
    {
      auto& data = *fields;
      auto& traversals = *Traversals;
      LBMGrid& Grid = domain->m_grid;

      // fill grid size;
      constexpr Area L = Area::Local;
      constexpr Traversal R = Traversal::All;
      //constexpr Traversal R = Traversal::Real;
      auto br = Grid.build_box<L, R>();
      onika::math::IJK local_grid_size(br.get_length(0), br.get_length(1), br.get_length(2));

      // storage
      auto& bb = *bbmanager;
      bb.resize_data(*periodic, local_grid_size, domain->MPI_coord, domain->MPI_grid_size);

      // get fields
      FieldView<Q> pf = data.distributions();

      // for clarity
      constexpr int dim_x = 0;
      constexpr int dim_y = 1;
      constexpr int dim_z = 2;
      launcher<dim_x, Side::Left>(traversals, pf, bb);
      launcher<dim_x, Side::Right>(traversals, pf, bb);
      launcher<dim_y, Side::Left>(traversals, pf, bb);
      launcher<dim_y, Side::Right>(traversals, pf, bb);
      launcher<dim_z, Side::Left>(traversals, pf, bb);
      launcher<dim_z, Side::Right>(traversals, pf, bb);
    }
  };

  // === register factories ===  
  ONIKA_AUTORUN_INIT(pre_bounce_back)
  {
    OperatorNodeFactory::instance()->register_factory( "pre_bounce_back", make_variant_operator<PreBounceBack>);
  }
}
