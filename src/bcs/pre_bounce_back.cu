#include "hip/hip_runtime.h"
#include <mpi.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <onika/log.h>
#include <onika/cuda/hip/hip_runtime.h>
#include <onika/memory/allocator.h>
#include <onika/parallel/parallel_for.h>

#include <onika/math/basic_types.h>
#include <grid_lbm/domain_lbm.hpp>
#include <grid_lbm/comm.hpp>
#include <grid_lbm/enum.hpp>
#include <grid_lbm/grid_data_lbm.hpp>
#include <grid_lbm/parallel_for_core.cu>
#include <grid_lbm/traversal_lbm.hpp>
#include <hippoLBM/bcs/bounce_back.hpp>
#include <hippoLBM/bcs/bounce_back_manager.hpp>

namespace hippoLBM
{
  using namespace onika;
  using namespace scg;
  using namespace onika::cuda;
  using BoolVector = std::vector<bool>;

  template<int Q>
    class PreBounceBack : public OperatorNode
  {
    ADD_SLOT( grid_data_lbm<Q>, GridDataQ, INPUT_OUTPUT, REQUIRED, DocString{"Grid data for the LBM simulation, including distribution functions and macroscopic fields."});
    ADD_SLOT( traversal_lbm, Traversals, INPUT, REQUIRED, DocString{"It contains different sets of indexes categorizing the grid points into Real, Edge, or All."});
    ADD_SLOT( domain_lbm<Q>, DomainQ, INPUT, REQUIRED);
    ADD_SLOT( bounce_back_manager<Q>, bbmanager, INPUT_OUTPUT);
    ADD_SLOT( BoolVector, periodic   , INPUT , REQUIRED );
    public:
    inline std::string documentation() const override final
    {
      return R"EOF( 
        )EOF";
    }

    template<int dim, Direction dir> 
      void launcher(traversal_lbm& traversals, WrapperF<Q>& pf, bounce_back_manager<Q>& bbm)
      {
        int idx = helper_dim_idx<dim,dir>();
        WrapperF<bounce_back_manager<Q>::Un> pfi = bbm.get_data(idx);
        if( pfi.N > 0 )
        {
          constexpr Traversal Tr = get_traversal<dim, dir>();
          auto [ptr, size] = traversals.get_data<Tr>();
          assert(size == size_t(pfi.N));
          assert(ptr != nullptr);

          ParallelForOptions opts;
          opts.omp_scheduling = OMP_SCHED_STATIC;
          pre_bounce_back<dim, dir, Q> kernel = {ptr};
          auto params = make_tuple(pf, pfi);
          parallel_for_id_runner runner = {kernel, params};
          parallel_for(size, runner, parallel_execution_context(), opts);
        }
      }

    inline void execute () override final
    {
      auto& data = *GridDataQ;
      auto& traversals = *Traversals;
      auto& domain = *DomainQ;
      grid<3>& Grid = domain.m_grid;

      // fill grid size;
      constexpr Area L = Area::Local;
      constexpr Traversal R = Traversal::Real;
      auto br = Grid.build_box<L, R>();
      onika::math::IJK local_grid_size(br.get_length(0), br.get_length(1), br.get_length(2));

      // storage
      auto& bb = *bbmanager;
      bb.resize_data(*periodic, local_grid_size, domain.MPI_coord, domain.MPI_grid_size);

      // get fields
      WrapperF<Q> pf = data.distributions();

      // for clarity
      constexpr int dim_x = 0;
      constexpr int dim_y = 1;
      constexpr int dim_z = 2;
      launcher<dim_x, Direction::Left>(traversals, pf, bb);
      launcher<dim_x, Direction::Right>(traversals, pf, bb);
      launcher<dim_y, Direction::Left>(traversals, pf, bb);
      launcher<dim_y, Direction::Right>(traversals, pf, bb);
      launcher<dim_z, Direction::Left>(traversals, pf, bb);
      launcher<dim_z, Direction::Right>(traversals, pf, bb);
    }
  };

  using PreBounceBack3D19Q = PreBounceBack<19>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT()
  {
    OperatorNodeFactory::instance()->register_factory( "pre_bounce_back", make_compatible_operator<PreBounceBack3D19Q>);
  }
}
